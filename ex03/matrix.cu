
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 1


// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);
// Part 2 of 4: implement the kernel
__global__ void kernel( int *a, int dimx, int dimy ) {

int idx_x = threadIdx.x + blockIdx.x * blockDim.x;
int idx_y = threadIdx.y + blockIdx.y * blockDim.y;

if ((idx_x < dimx) && (idx_y < dimy)) {
a[idx_y * dimx + idx_x] = idx_y * dimx + idx_x;
}
} 


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main() {
    hipSetDevice(MYDEVICE);
// Part 1 and 4 of 4: set the dimensions of the matrix
    int dimx = 19;
    int dimy = 67;
    int num_bytes = dimx*dimy*sizeof(int);

    int *d_a=0, *h_a=0; // device and host pointers

    h_a = (int*)malloc(num_bytes);
    //allocate memory on the device
    hipMalloc(&d_a, num_bytes);

    if( NULL==h_a || NULL==d_a ) {
        std::cerr << "couldn't allocate memory" << std::endl;
        return 1;
    }

    // Part 2 of 4: define grid and block size and launch the kernel
    dim3 grid, block;
    block.x = 20;
    block.y = 20;
    grid.x  = (dimx / block.x) + 1;
    grid.y  = (dimy / block.y) + 1;

    // host to device copy
    hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);

    kernel<<<grid, block>>>( d_a, dimx, dimy );
    // block until the device has completed
    hipDeviceSynchronize();
    // check if kernel execution generated an error
    checkCUDAError("kernel execution");
    // device to host copy
    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");
    // verify the data returned to the host is correct
    for(int row=0; row<dimy; row++)
    {
        for(int col=0; col<dimx; col++){
	    if (h_a[row * dimx + col] != row * dimx + col) {
		std::cout<<h_a[row * dimx + col] << std::endl;
		std::cout<<(row * dimx + col) << std::endl;
		}
            assert(h_a[row * dimx + col] == row * dimx + col);
	}
    }
    // free host memory
    free( h_a );
    // free device memory
    hipFree( d_a );

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    std::cout << "Correct!" << std::endl;

    return 0;
}


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }                         
}
